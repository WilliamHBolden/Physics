#include "hip/hip_runtime.h"
#include "Physics.cuh"
//#include "Math.cuh"
#include "stdio.h"

#define G 6.67e-11f
#define k 7.0e0f  //7.0e-1f
#define da 10.0f  //0.9f

#define k2 1.0e2f


__global__ void nBody(Body* bodies, int numBodies, float ks) //would also pass physics model
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < numBodies)
	{
		bodies[i].force = glm::vec3(0,0,0);
		glm::vec3 fGravity(0,0,0);

		for(int n = 0; n < numBodies; n++)
		{
			if(n != i)
			{
				glm::vec3 rin = bodies[n].position - bodies[i].position; 

				float dotr = glm::dot(rin, rin);
				float distr = sqrtf(dotr);

				glm::vec3 dirvec = rin/distr;

				if(distr < 2)
				fGravity += bodies[n].mass * dirvec*0.25f;
				else
				fGravity += bodies[n].mass * dirvec/ (distr*distr);

				
				if(distr < 2)
				{
					glm::vec3 colp = bodies[i].position + 0.5f*rin;

					glm::vec3 colforce = -ks* (2 - distr)*dirvec;

					glm::vec3 aveli = bodies[i].invITensor * bodies[i].aMomentum;
					glm::vec3 aveln = bodies[n].invITensor * bodies[n].aMomentum;

					glm::vec3 veli = bodies[i].lMomentum/bodies[i].mass;
					glm::vec3 veln = bodies[n].lMomentum/bodies[n].mass;

					glm::vec3 relPosCM = 0.5f * rin; //reltive position of collision to center of mass of i

					glm::vec3 tveli(0, 0, 0);
					/*
					tveli.x += aveli.x*relPosCM.x;
					tveli.y += aveli.y*relPosCM.y;
					tveli.z += aveli.z*relPosCM.z;
					*/
					tveli += veli;
					

					// - for radius
					glm::vec3 tveln(0, 0, 0);
					/*
					tveln.x += -aveln.x*relPosCM.x;
					tveln.y += -aveln.y*relPosCM.y;
					tveln.z += -aveln.z*relPosCM.z;
					*/
					tveln += veln;
					

					glm::vec3 tvel= tveli - tveln;

					glm::vec3 shearforce = k * tvel;

					glm::vec3 damp = tvel * -da;
					//glm::vec3 damp = tvel * -0.7f;

					bodies[i].force += (colforce + shearforce + damp);


				//	bodies[i].torque += glm::cross(relPosCM, (colforce + shearforce + damp));
				}
			}
		}

		fGravity = fGravity * G * bodies[i].mass;
		bodies[i].force += fGravity;
		
	}
}

//__global__ void 
__global__ void integrate(Body* bodies, glm::mat4* models, glm::vec4* colors, int numBodies ,float timestep, float boost)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < numBodies)
	{
		//bodies[i].force += 0.0001;

		/*
			should i include damping?
		*/


		/*
			translation
		*/
		bodies[i].lMomentum += bodies[i].force * timestep;
		glm::vec3 vel = bodies[i].lMomentum/bodies[i].mass;
		bodies[i].position += vel * timestep;

		/*
			Rotation
		*/
		//update aMom
		bodies[i].aMomentum += bodies[i].torque * timestep;

		//Update orientation

		glm::vec3 avel = bodies[i].invITensor * bodies[i].aMomentum;
		avel *= timestep;
		
		float angle = glm::length(avel);

		if(angle > 0)
		{
			float sinal = sinf(angle/2.0f)/angle;
			glm::quat inter(cosf(angle/2.0f) ,glm::vec3(avel.x*sinal, avel.y*sinal, avel.z*sinal));
			bodies[i].orienation = inter * bodies[i].orienation;
		}

		glm::mat4 modelMat = glm::mat4_cast(bodies[i].orienation);

		modelMat[3][0] += bodies[i].position.x;
		modelMat[3][1] += bodies[i].position.y;
		modelMat[3][2] += bodies[i].position.z;

		models[i] = modelMat;


		/*
			Coloring
		*/
		colors[i] = glm::vec4(bodies[i].force* 10.5f/boost, 1.0f) + glm::vec4(0.3f, 0.3f, 0.3f, 0.0f);
	}
}

__global__ void test(Body* bodies, glm::vec4* colors, int numBodies)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < numBodies)
	{
		//printf("%f, %f, %f, %f\n", colors[i].x, colors[i].y, colors[i].z, colors[i].w);

		printf("%f, %f, %f\n", bodies[i].force.x, bodies[i].force.y, bodies[i].force.z);
	}
}



void runPhysics(Body* bodies, glm::mat4* models, glm::vec4* colors, int numBodies, float timestep, float boost)
{
	dim3 blockSize = 512;
	dim3 gridSize = dim3((numBodies+blockSize.x-1)/blockSize.x);

//	printf("%d, %d\n", sizeof(glm::mat4), sizeof(float)*16);


	nBody<<<gridSize, blockSize>>>(bodies, numBodies, k * boost);

	integrate<<<gridSize, blockSize>>>(bodies, models, colors, numBodies, timestep, boost);
	//test<<<gridSize, blockSize>>>(bodies, colors, numBodies);


	hipDeviceSynchronize();
}

