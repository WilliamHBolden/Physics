#include "hip/hip_runtime.h"
#include "Physics.cuh"
#include "stdio.h"


__global__ void nBody(Body* bodies, int numBodies) //would also pass physics model
{

}

//__global__ void 
__global__ void integrate(Body* bodies, glm::mat4* models, glm::vec4* colors, int numBodies ,float timestep)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < numBodies)
	{
		colors[i] = glm::vec4(1.0f, 1.0f, 1.0f, 1.0f);
	}
}

__global__ void test(glm::mat4* models, glm::vec4* colors, int numBodies)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < numBodies)
	{
		printf("%f, %f, %f, %f\n", colors[i].x, colors[i].y, colors[i].z, colors[i].w);
	}
}

void runPhysics(Body* bodies, glm::mat4* models, glm::vec4* colors, int numBodies, float timestep)
{
	dim3 blockSize = 512;
	dim3 gridSize = dim3((numBodies+blockSize.x-1)/blockSize.x);

//	printf("%d, %d\n", sizeof(glm::mat4), sizeof(float)*16);

//	nBody<<<gridSize, blockSize>>>(bodies);
	integrate<<<gridSize, blockSize>>>(bodies, models, colors, numBodies, timestep);
	//test<<<gridSize, blockSize>>>(models, colors, numBodies);



}

